#include "hip/hip_runtime.h"
#include "intersections.h"

__host__ __device__ float boxIntersectionTest(
    Geom box,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    Ray q;
    q.origin    =                multiplyMV(box.inverseTransform, glm::vec4(r.origin   , 1.0f));
    q.direction = glm::normalize(multiplyMV(box.inverseTransform, glm::vec4(r.direction, 0.0f)));

    float tmin = -1e38f;
    float tmax = 1e38f;
    glm::vec3 tmin_n;
    glm::vec3 tmax_n;
    for (int xyz = 0; xyz < 3; ++xyz)
    {
        float qdxyz = q.direction[xyz];
        /*if (glm::abs(qdxyz) > 0.00001f)*/
        {
            float t1 = (-0.5f - q.origin[xyz]) / qdxyz;
            float t2 = (+0.5f - q.origin[xyz]) / qdxyz;
            float ta = glm::min(t1, t2);
            float tb = glm::max(t1, t2);
            glm::vec3 n;
            n[xyz] = t2 < t1 ? +1 : -1;
            if (ta > 0 && ta > tmin)
            {
                tmin = ta;
                tmin_n = n;
            }
            if (tb < tmax)
            {
                tmax = tb;
                tmax_n = n;
            }
        }
    }

    if (tmax >= tmin && tmax > 0)
    {
        outside = true;
        if (tmin <= 0)
        {
            tmin = tmax;
            tmin_n = tmax_n;
            outside = false;
        }
        intersectionPoint = multiplyMV(box.transform, glm::vec4(getPointOnRay(q, tmin), 1.0f));
        normal = glm::normalize(multiplyMV(box.invTranspose, glm::vec4(tmin_n, 0.0f)));
        return glm::length(r.origin - intersectionPoint);
    }

    return -1;
}

__host__ __device__ float sphereIntersectionTest(
    Geom sphere,
    Ray r,
    glm::vec3 &intersectionPoint,
    glm::vec3 &normal,
    bool &outside)
{
    float radius = .5;

    glm::vec3 ro = multiplyMV(sphere.inverseTransform, glm::vec4(r.origin, 1.0f));
    glm::vec3 rd = glm::normalize(multiplyMV(sphere.inverseTransform, glm::vec4(r.direction, 0.0f)));

    Ray rt;
    rt.origin = ro;
    rt.direction = rd;

    float vDotDirection = glm::dot(rt.origin, rt.direction);
    float radicand = vDotDirection * vDotDirection - (glm::dot(rt.origin, rt.origin) - powf(radius, 2));
    if (radicand < 0)
    {
        return -1;
    }

    float squareRoot = sqrt(radicand);
    float firstTerm = -vDotDirection;
    float t1 = firstTerm + squareRoot;
    float t2 = firstTerm - squareRoot;

    float t = 0;
    if (t1 < 0 && t2 < 0)
    {
        return -1;
    }
    else if (t1 > 0 && t2 > 0)
    {
        t = min(t1, t2);
        outside = true;
    }
    else
    {
        t = max(t1, t2);
        outside = false;
    }

    glm::vec3 objspaceIntersection = getPointOnRay(rt, t);

    intersectionPoint = multiplyMV(sphere.transform, glm::vec4(objspaceIntersection, 1.f));
    normal = glm::normalize(multiplyMV(sphere.invTranspose, glm::vec4(objspaceIntersection, 0.f)));
    if (!outside)
    {
        normal = -normal;
    }

    return glm::length(r.origin - intersectionPoint);
}

__host__ __device__ glm::vec3 barycentric(glm::vec3 p, glm::vec3 t1, glm::vec3 t2, glm::vec3 t3) {
    glm::vec3 edge1 = t2 - t1;
    glm::vec3 edge2 = t3 - t2;
    float S = length(cross(edge1, edge2));
    edge1 = p - t2;
    edge2 = p - t3;
    float S1 = length(cross(edge1, edge2));
    edge1 = p - t1;
    edge2 = p - t3;
    float S2 = length(cross(edge1, edge2));
    edge1 = p - t1;
    edge2 = p - t2;
    float S3 = length(cross(edge1, edge2));
    return glm::vec3(S1 / S, S2 / S, S3 / S);
}

__host__ __device__ float triangleIntersectionTest(
	glm::vec3 v0, glm::vec3 v1, glm::vec3 v2,
    Geom geom, Triangle tri,
	Ray r,
	glm::vec3& intersectionPoint,
	glm::vec3& normal,
    glm::vec3& uv,
    glm::mat3& TBN,
	bool& outside)
{
    glm::vec3 ro = multiplyMV(geom.inverseTransform, glm::vec4(r.origin, 1.0f));
    glm::vec3 rd = glm::normalize(multiplyMV(geom.inverseTransform, glm::vec4(r.direction, 0.0f)));

    Ray rt;
    rt.origin = ro;
    rt.direction = rd;

	glm::vec3 e1 = v1 - v0;
	glm::vec3 e2 = v2 - v0;
    glm::vec3 deltaUV1 = tri.uvs[1] - tri.uvs[0];
    glm::vec3 deltaUV2 = tri.uvs[2] - tri.uvs[0];
	glm::vec3 h = cross(rt.direction, e2);
    float a = glm::dot(e1, h);

	if (a > -0.00001f && a < 0.00001f) // parallel
	{
		return -1;
	}

	float f = 1.0f / a;
	glm::vec3 s = rt.origin - v0;
	float u = f * glm::dot(s, h);
	if (u < 0.0f || u > 1.0f)
	{
		return -1;
	}
	glm::vec3 q = cross(s, e1);
	float v = f * glm::dot(rt.direction, q);
	if (v < 0.0f || u + v > 1.0f)
	{
		return -1;
	}

	float t = f * glm::dot(e2, q);
    float f2 = 1.0f / (deltaUV1.x * deltaUV2.y - deltaUV2.x * deltaUV1.y);
	if (t > 0.00001f)
	{
		glm::vec3 objspaceIntersection = getPointOnRay(rt, t);
		intersectionPoint = multiplyMV(geom.transform, glm::vec4(objspaceIntersection, 1.f));
        normal = glm::normalize(multiplyMV(geom.invTranspose, glm::vec4(objspaceIntersection, 0.f)));
        glm::vec3 bary = barycentric(objspaceIntersection, v0, v1, v2);
        glm::vec3 triUV0 = tri.uvs[0];
        glm::vec3 triUV1 = tri.uvs[1];
        glm::vec3 triUV2 = tri.uvs[2];
        uv = bary.x * tri.uvs[0] + bary.y * tri.uvs[1] + bary.z * tri.uvs[2];
        // compute TBN
        glm::vec3 tangent;
        tangent.x = f * (deltaUV2.y * e1.x - deltaUV1.y * e2.x);
        tangent.y = f * (deltaUV2.y * e1.y - deltaUV1.y * e2.y);
        tangent.z = f * (deltaUV2.y * e1.z - deltaUV1.y * e2.z);
        tangent = glm::normalize(tangent);
        glm::vec3 bitangent;
        bitangent.x = f * (-deltaUV2.x * e1.x + deltaUV1.x * e2.x);
        bitangent.y = f * (-deltaUV2.x * e1.y + deltaUV1.x * e2.y);
        bitangent.z = f * (-deltaUV2.x * e1.z + deltaUV1.x * e2.z);
        bitangent = glm::normalize(bitangent);
        // Gram-Schmidt to orthogonalize
        tangent = glm::normalize(tangent - glm::dot(tangent, normal) * normal);
        bitangent = glm::cross(normal, tangent);
        bitangent = glm::normalize(bitangent);
        glm::mat3 TBN = glm::mat3(tangent, bitangent, normal);
		outside = true;
		return glm::length(r.origin - intersectionPoint);
	}
    else
	{
		return -1;
	}

}

__host__ __device__ float meshIntersectionTest(
    Geom geom,
    Ray r,
    glm::vec3& intersectionPoint,
    glm::vec3& normal,
    glm::vec3& uv,
    glm::mat3& TBN,
    bool& outside
) {
    float closestT = FLT_MAX;
    int hitTriangle = -1;
    int num_triangles = geom.mesh->num_triangles;
	Mesh* mesh = geom.mesh;

    for (int i = 0; i < num_triangles; ++i) {
        const Triangle& tri = geom.mesh->triangles[i];
        glm::vec3 tempIntersectionPoint, tempNormal, tempUV;
        glm::mat3 tempTBN;
        bool tempOutside;

        glm::vec3 p0 = tri.points[0];
		float p0x = p0.x;
		float p0y = p0.y;
		float p0z = p0.z;

		glm::vec3 p1 = tri.points[1];
		glm::vec3 p2 = tri.points[2];

        float t = 0.0;
        
        t = triangleIntersectionTest(
            tri.points[0], tri.points[1], tri.points[2],
            geom, tri,
            r,
            tempIntersectionPoint,
            tempNormal,
            tempUV,
            tempTBN,
            tempOutside
        );
        

        if (t > 0 && t < closestT) {
            closestT = t;
            intersectionPoint = tempIntersectionPoint;
            normal = tempNormal;
            outside = tempOutside;
            hitTriangle = i;
        }
    }

    return (hitTriangle != -1) ? closestT : -1.0f;
}
